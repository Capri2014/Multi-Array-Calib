#include "hip/hip_runtime.h"
/*Small program to quickly get probablity as matlab cant loop to save itself
 * call using logpdfT(R,vR,tA,vtA,vB,vtB,RB,vRB,t)
 */ 

#include "logpdfTG.h"

__device__ void V2R(const double* const vect, double* const R){
    
    double s,c,k,x,y,z;
    
    //get rotation mag
    double m = sqrt(vect[0]*vect[0] + vect[1]*vect[1] + vect[2]*vect[2]);
    
    //check for zero case and set angles
    if( m < 1.0e-12){
        s = 0;
        c = 1;
        k = 0;
        x = 0;
        y = 0;
        z = 0;
    }
    else{
        s = sin(m);
        c = cos(m);
        k = 1 - c;
        x = vect[0]/m;
        y = vect[1]/m;
        z = vect[2]/m;
    }
	
    //get rotation matrix
     R[0] = k*x*x+c;
     R[1] = k*x*y + s*z;
     R[2] = k*x*z - s*y;
     R[3] = k*x*y - s*z;
     R[4] = k*y*y + c;
     R[5] = k*y*z + s*x;
     R[6] = k*x*z + s*y;
     R[7] = k*y*z - s*x;
     R[8] = k*z*z + c;
}

__device__ void findScaleB(double* sB, double* tA, double* tB, double* RA, double* RB, double* Rs, double* Re, double* t){

    double RAM[9];
    double RBM[9];
    double RSM[9];
    double REM[9];
    double RM[9];
    
    V2R(RA,RAM);
    V2R(RB,RBM);
    V2R(Rs,RSM);
    V2R(Re,REM);
    
    RM[0] = RSM[0]*REM[0] + RSM[1]*REM[1] + RSM[2]*REM[2];
    RM[1] = RSM[3]*REM[0] + RSM[4]*REM[1] + RSM[5]*REM[2];
    RM[2] = RSM[6]*REM[0] + RSM[7]*REM[1] + RSM[8]*REM[2];
    RM[3] = RSM[0]*REM[3] + RSM[1]*REM[4] + RSM[2]*REM[5];
    RM[4] = RSM[3]*REM[3] + RSM[4]*REM[4] + RSM[5]*REM[5];
    RM[5] = RSM[6]*REM[3] + RSM[7]*REM[4] + RSM[8]*REM[5];
    RM[6] = RSM[0]*REM[6] + RSM[1]*REM[7] + RSM[2]*REM[8];
    RM[7] = RSM[3]*REM[6] + RSM[4]*REM[7] + RSM[5]*REM[8];
    RM[8] = RSM[6]*REM[6] + RSM[7]*REM[7] + RSM[8]*REM[8];
    
    double x[3];
    x[0] = -RM[0]*t[0] - RM[1]*t[1] - RM[2]*t[2];
    x[1] = -RM[3]*t[0] - RM[4]*t[1] - RM[5]*t[2];
    x[2] = -RM[6]*t[0] - RM[7]*t[1] - RM[8]*t[2];
    
    sB[0] = (RAM[0]*t[0] + RAM[3]*t[1] + RAM[6]*t[2] - t[0] + tA[0]) / (RM[0]*tB[0] + RM[3]*tB[1] + RM[6]*tB[2]);
    sB[1] = (RAM[1]*t[0] + RAM[4]*t[1] + RAM[7]*t[2] - t[1] + tA[1]) / (RM[1]*tB[0] + RM[4]*tB[1] + RM[7]*tB[2]);
    sB[2] = (RAM[2]*t[0] + RAM[5]*t[1] + RAM[8]*t[2] - t[2] + tA[2]) / (RM[2]*tB[0] + RM[5]*tB[1] + RM[8]*tB[2]);
    
    sB[3] = -(RBM[0]*x[0] + RBM[3]*x[1] + RBM[6]*x[2] - x[0] - RM[0]*tA[0] - RM[1]*tA[1] - RM[2]*tA[2]) / tB[0];
    sB[4] = -(RBM[1]*x[0] + RBM[4]*x[1] + RBM[7]*x[2] - x[1] - RM[3]*tA[0] - RM[4]*tA[1] - RM[5]*tA[2]) / tB[1];
    sB[5] = -(RBM[2]*x[0] + RBM[5]*x[1] + RBM[8]*x[2] - x[2] - RM[6]*tA[0] - RM[7]*tA[1] - RM[8]*tA[2]) / tB[2];
}

__device__ void scaleElementVarB(double* err, double* base, double v, double* tA, double* tB, double* RA, double* RB, double* Rs, double* Re, double* t){
    double temp[6];
    
    findScaleB(temp,tA,tB,RA,RB,Rs,Re,t);
    for(size_t i = 0; i < 6; i++){
        err[i] += v*(temp[i] - base[i])*(temp[i] - base[i]);
    }
}

__global__ void  combineScaleB(double* stBPtr, double* svtBPtr,
        double* tAPtr, double* vtAPtr,
        double* tBPtr, double* vtBPtr,
        double* RAPtr, double* vRAPtr,
        double* RBPtr, double* vRBPtr,
        double* RPtr, double* vRPtr,
        double* t, size_t n){
    
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= n){
		return;
	}
    
    double* tA = &tAPtr[3*i];
    double* tB = &tBPtr[3*i];
    double* RA = &RAPtr[3*i];
    double* RB = &RBPtr[3*i];
    double* vtA = &vtAPtr[3*i];
    double* vtB = &vtBPtr[3*i];
    double* vRA = &vRAPtr[3*i];
    double* vRB = &vRBPtr[3*i];

	double* stB = &stBPtr[3 * i];
	double* svtB = &svtBPtr[3 * i];

    double* Rs = &RPtr[0];
    double* Re = &RPtr[3];
    double* vRs = &vRPtr[0];
    double* vRe = &vRPtr[3];
    
    double base[6];
    
    findScaleB(base, tA, tB, RA, RB, Rs, Re, t);
    
    //add OFFSET to estimate variance
    double err[] = {0,0,0,0,0,0};

    for(size_t j = 0; j < 3; j++){
        tA[j] += OFFSET;
        scaleElementVarB(err,base, vtA[j], tA, tB, RA, RB, Rs, Re, t);
        tA[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        tB[j] += OFFSET;
        scaleElementVarB(err,base, vtB[j], tA, tB, RA, RB, Rs, Re, t);
        tB[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        RA[j] += OFFSET;
        scaleElementVarB(err,base, vRA[j], tA, tB, RA, RB, Rs, Re, t);
        RA[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        RB[j] += OFFSET;
        scaleElementVarB(err,base, vRB[j], tA, tB, RA, RB, Rs, Re, t);
        RB[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        Rs[j] += OFFSET;
        scaleElementVarB(err,base, vRs[j], tA, tB, RA, RB, Rs, Re, t);
        Rs[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        Re[j] += OFFSET;
        scaleElementVarB(err,base, vRe[j], tA, tB, RA, RB, Rs, Re, t);
        Re[j] -= OFFSET;
    }
    
    //transform to variance
    for(size_t j = 0; j < 6; j++){
        err[j] = err[j]/(OFFSET*OFFSET);
        err[j] = 2/err[j];//2 as equations depend on each other alot
    }
    
    //combine estimates
    double s = 0;
    double sV = 0;
    for(size_t j = 0; j < 6; j++){
        s += base[j]*err[j];
        sV += err[j];
    }
    sV = (1/sV); 
    s = s*sV;
        
    //add to original
    for(size_t j = 0; j< 3; j++){
        stB[j] = s*tB[j];
        svtB[j] = s*s*vtB[j] + sV*tB[j]*tB[j];
    }
}

__device__ void findError(double* err, double* tA, double* tB, double* RA, double* RB, double* Rs, double* Re, double* t){
    
    double RAM[9];
    double RBM[9];
    double RSM[9];
    double REM[9];
    double RM[9];
    
    V2R(RA,RAM);
    V2R(RB,RBM);
    V2R(Rs,RSM);
    V2R(Re,REM);
    
    RM[0] = RSM[0]*REM[0] + RSM[1]*REM[1] + RSM[2]*REM[2];
    RM[1] = RSM[3]*REM[0] + RSM[4]*REM[1] + RSM[5]*REM[2];
    RM[2] = RSM[6]*REM[0] + RSM[7]*REM[1] + RSM[8]*REM[2];
    RM[3] = RSM[0]*REM[3] + RSM[1]*REM[4] + RSM[2]*REM[5];
    RM[4] = RSM[3]*REM[3] + RSM[4]*REM[4] + RSM[5]*REM[5];
    RM[5] = RSM[6]*REM[3] + RSM[7]*REM[4] + RSM[8]*REM[5];
    RM[6] = RSM[0]*REM[6] + RSM[1]*REM[7] + RSM[2]*REM[8];
    RM[7] = RSM[3]*REM[6] + RSM[4]*REM[7] + RSM[5]*REM[8];
    RM[8] = RSM[6]*REM[6] + RSM[7]*REM[7] + RSM[8]*REM[8];
        
    double x[3];
    x[0] = -RM[0]*t[0] - RM[1]*t[1] - RM[2]*t[2];
    x[1] = -RM[3]*t[0] - RM[4]*t[1] - RM[5]*t[2];
    x[2] = -RM[6]*t[0] - RM[7]*t[1] - RM[8]*t[2];
    
    err[0] = RAM[0]*t[0] + RAM[3]*t[1] + RAM[6]*t[2] - t[0] - RM[0]*tB[0] - RM[3]*tB[1] - RM[6]*tB[2] + tA[0];
    err[1] = RAM[1]*t[0] + RAM[4]*t[1] + RAM[7]*t[2] - t[1] - RM[1]*tB[0] - RM[4]*tB[1] - RM[7]*tB[2] + tA[1];
    err[2] = RAM[2]*t[0] + RAM[5]*t[1] + RAM[8]*t[2] - t[2] - RM[2]*tB[0] - RM[5]*tB[1] - RM[8]*tB[2] + tA[2];
    
    err[3] = RBM[0]*x[0] + RBM[3]*x[1] + RBM[6]*x[2] - x[0] - RM[0]*tA[0] - RM[1]*tA[1] - RM[2]*tA[2] + tB[0];
    err[4] = RBM[1]*x[0] + RBM[4]*x[1] + RBM[7]*x[2] - x[1] - RM[3]*tA[0] - RM[4]*tA[1] - RM[5]*tA[2] + tB[1];
    err[5] = RBM[2]*x[0] + RBM[5]*x[1] + RBM[8]*x[2] - x[2] - RM[6]*tA[0] - RM[7]*tA[1] - RM[8]*tA[2] + tB[2];
} 

__device__ void errorElementVar(double* err, double* base, double v, double* tA, double* tB, double* RA, double* RB, double* Rs, double* Re, double* t){
    double temp[6];
        
    findError(temp,tA,tB,RA,RB,Rs,Re,t);
    for(size_t i = 0; i < 6; i++){
        err[i] += v*(temp[i] - base[i])*(temp[i] - base[i]);
    }
}

__global__ void findErrVar(double* loglPtr,
        double* tAPtr, double* vtAPtr,
        double* tBPtr, double* vtBPtr,
        double* RAPtr, double* vRAPtr,
        double* RBPtr, double* vRBPtr,
        double* RPtr, double* vRPtr,
        double* t, size_t n){
    
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= n){
		return;
	}
    
    double* tA = &tAPtr[3*i];
    double* tB = &tBPtr[3*i];
    double* RA = &RAPtr[3*i];
    double* RB = &RBPtr[3*i];
    double* vtA = &vtAPtr[3*i];
    double* vtB = &vtBPtr[3*i];
    double* vRA = &vRAPtr[3*i];
    double* vRB = &vRBPtr[3*i];

    double* Rs = &RPtr[0];
    double* Re = &RPtr[3];
    double* vRs = &vRPtr[0];
    double* vRe = &vRPtr[3];
    
    double* logl = &loglPtr[i];
    
    double err[6];
    double verr[6];
    
    findError(err, tA, tB, RA, RB, Rs, Re, t);
    
    //add OFFSET to estimate variance
    for(size_t j = 0; j < 6; j++){
        verr[j] = 0;
    }
    
    for(size_t j = 0; j < 3; j++){
        tA[j] += OFFSET;
        errorElementVar(verr,err, vtA[j], tA, tB, RA, RB, Rs, Re, t);
        tA[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        tB[j] += OFFSET;
        errorElementVar(verr,err, vtB[j], tA, tB, RA, RB, Rs, Re, t);
        tB[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        RA[j] += OFFSET;
        errorElementVar(verr,err, vRA[j], tA, tB, RA, RB, Rs, Re, t);
        RA[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        RB[j] += OFFSET;
        errorElementVar(verr,err, vRB[j], tA, tB, RA, RB, Rs, Re, t);
        RB[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        Rs[j] += OFFSET;
        errorElementVar(verr,err, vRs[j], tA, tB, RA, RB, Rs, Re, t);
        Rs[j] -= OFFSET;
    }
    for(size_t j = 0; j < 3; j++){
        Re[j] += OFFSET;
        errorElementVar(verr,err, vRe[j], tA, tB, RA, RB, Rs, Re, t);
        Re[j] -= OFFSET;
    }
    
    //transform to variance
    for(size_t j = 0; j < 6; j++){
        verr[j] = verr[j]/(OFFSET*OFFSET);
    }
    
    //find exponential exponent
    double eExp1 = -0.5*(err[0]*err[0]/verr[0] + err[1]*err[1]/verr[1] + err[2]*err[2]/verr[2]);
    //find part before exponential
    double bExp1 = -log(sqrt(8*PI*PI*PI*verr[0]*verr[1]*verr[2]));

    //find exponential exponent
    double eExp2 = -0.5*(err[3]*err[3]/verr[3] + err[4]*err[4]/verr[4] + err[5]*err[5]/verr[5]);
    //find part before exponential
    double bExp2 = -log(sqrt(8*PI*PI*PI*verr[3]*verr[4]*verr[5]));

    //finding log likelihood
    logl[0] = (bExp1 + eExp1 + bExp2 + eExp2)/2;
}  

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    
    //check inputs
    if (nrhs != 12 || (nlhs != 1 && nlhs != 0)) {
      mexErrMsgIdAndTxt("CPROB:BadNArgs", 
                        "Need 12 inputs and 1 output.");
    }
    
    //get value of input variables
    mxGPUArray const * RIn = mxGPUCreateFromMxArray(prhs[0]);
    double* RPtr = (double*)(mxGPUGetDataReadOnly(RIn));
    mxGPUArray const * vRIn = mxGPUCreateFromMxArray(prhs[1]);
    double* vRPtr = (double*)(mxGPUGetDataReadOnly(vRIn));
    mxGPUArray const * tAIn = mxGPUCreateFromMxArray(prhs[2]);
    double* tAPtr = (double*)(mxGPUGetDataReadOnly(tAIn));
    mxGPUArray const * vtAIn = mxGPUCreateFromMxArray(prhs[3]);
    double* vtAPtr = (double*)(mxGPUGetDataReadOnly(vtAIn));
    mxGPUArray const * RAIn = mxGPUCreateFromMxArray(prhs[4]);
    double* RAPtr = (double*)(mxGPUGetDataReadOnly(RAIn));
    mxGPUArray const * vRAIn = mxGPUCreateFromMxArray(prhs[5]);
    double* vRAPtr = (double*)(mxGPUGetDataReadOnly(vRAIn));
    mxGPUArray const * tBIn = mxGPUCreateFromMxArray(prhs[6]);
    double* tBPtr = (double*)(mxGPUGetDataReadOnly(tBIn));
    mxGPUArray const * vtBIn = mxGPUCreateFromMxArray(prhs[7]);
    double* vtBPtr = (double*)(mxGPUGetDataReadOnly(vtBIn));
    mxGPUArray const * RBIn = mxGPUCreateFromMxArray(prhs[8]);
    double* RBPtr = (double*)(mxGPUGetDataReadOnly(RBIn));
    mxGPUArray const * vRBIn = mxGPUCreateFromMxArray(prhs[9]);
    double* vRBPtr = (double*)(mxGPUGetDataReadOnly(vRBIn));
    mxGPUArray const * tIn = mxGPUCreateFromMxArray(prhs[10]);
    double* tPtr = (double*)(mxGPUGetDataReadOnly(tIn));
    double * s = mxGetPr(prhs[11]);
    
    //gets size of variables
    const size_t n = mxGPUGetDimensions(tAIn)[1];
    
    //setup outputs
	mwSize outSize[] = {n,1};
    mxGPUArray* outMat = mxGPUCreateGPUArray(2, outSize, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	plhs[0] = mxGPUCreateMxArrayOnGPU(outMat);
    double* outPtr = (double*)(mxGPUGetData(outMat));
    
    //setup storage
    outSize[0] = 3;
    outSize[1] = n;
    mxGPUArray* stBIn = mxGPUCreateGPUArray(2, outSize, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double* stBPtr = (double*)(mxGPUGetData(stBIn));
    mxGPUArray* svtBIn = mxGPUCreateGPUArray(2, outSize, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    double* svtBPtr = (double*)(mxGPUGetData(svtBIn));
    
    if(s[0] != 0){
        combineScaleB<<<gridSize(n), BLOCK_SIZE>>>(stBPtr,svtBPtr,tAPtr,vtAPtr,tBPtr,vtBPtr,RAPtr,vRAPtr,RBPtr,vRBPtr,RPtr,vRPtr,tPtr,n);
        CudaCheckError();
        findErrVar<<<gridSize(n), BLOCK_SIZE>>>(outPtr,tAPtr,vtAPtr,stBPtr,svtBPtr,RAPtr,vRAPtr,RBPtr,vRBPtr,RPtr,vRPtr,tPtr,n);
        CudaCheckError();
    }
    else{
        findErrVar<<<gridSize(n), BLOCK_SIZE>>>(outPtr,tAPtr,vtAPtr,tBPtr,vtBPtr,RAPtr,vRAPtr,RBPtr,vRBPtr,RPtr,vRPtr,tPtr,n);
        CudaCheckError();
    }
    
	mxGPUDestroyGPUArray(RIn);
	mxGPUDestroyGPUArray(vRIn);
	mxGPUDestroyGPUArray(tAIn);
	mxGPUDestroyGPUArray(vtAIn);
	mxGPUDestroyGPUArray(RAIn);
	mxGPUDestroyGPUArray(vRAIn);
	mxGPUDestroyGPUArray(tBIn);
	mxGPUDestroyGPUArray(vtBIn);
	mxGPUDestroyGPUArray(RBIn);
	mxGPUDestroyGPUArray(vRBIn);
	mxGPUDestroyGPUArray(tIn);

    mxGPUDestroyGPUArray(stBIn);
    mxGPUDestroyGPUArray(svtBIn);
	mxGPUDestroyGPUArray(outMat);
}
